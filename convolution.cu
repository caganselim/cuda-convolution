#include "hip/hip_runtime.h"
/*
 * CUDA Convolution Library
 *
 * v1.0.0
 *
 * C.S. Coban
 *
 */

#include <stdio.h>
#include <string.h>

//Cuda Libs
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include ""

//OpenCV Libs
#include "opencv2/opencv.hpp"
#include "helperLibs/hip/hip_runtime_api.h"
#include "helperLibs/helper_functions.h"

//CUDA Error Checker
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

//Allocate filter kernel in constant memory on device
__constant__ int filter_kernel[] =
{
  1, 4, 6, 4, 1,
  4, 16, 24, 16, 4,
  6, 24, 36, 24, 6,
  4, 16, 24, 16, 4,
  1, 4, 6, 4, 1,
};

//Set block size
#define BLOCK_SIZE 16

/* KERNELS */


__global__ void conv_globalMem( unsigned char* input,
								unsigned char* output,
								int width,
								int height,
								int inputWidthStep,
								int outputWidthStep,
								int radius, int weight){

	int xIndex = blockIdx.x*blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y*blockDim.y + threadIdx.y;


	//Only valid threads perform memory I/O
	if((xIndex < width) && (yIndex < height)){

		//Allocate values
		int3 acc = make_int3(0,0,0);
		int3 val = make_int3(0,0,0);

		int output_tid = yIndex * outputWidthStep + (3 * xIndex);

		for (int i = -radius; i <= radius; i++) {
			for (int j = -radius; j <= radius; j++) {

				//Skip violations (which will lead to zero by default
				if ((xIndex + i < 0) || (xIndex + i >= width) || (yIndex + j < 0) || (yIndex + j >= height)) continue;

				//Get kernel value
				int temp = filter_kernel[i + radius + (j+radius)*((radius << 1) + 1)];

				//Location of colored pixel in input
				int input_tid = (yIndex + j) * inputWidthStep + (3 * (xIndex + i));

				//Fetch the three channel values
				const unsigned char blue	= input[input_tid];
				const unsigned char green = input[input_tid + 1];
				const unsigned char red	= input[input_tid + 2];


				val.x = int(blue)*temp;
				val.y = int(green)*temp;
				val.z = int(red)*temp;

				//Perform cumulative sum
				acc.x += val.x;
				acc.y += val.y;
				acc.z += val.z;
			}
		}

		acc.x = acc.x/weight;
		acc.y = acc.y/weight;
		acc.z = acc.z/weight;

		output[output_tid] = static_cast<unsigned char>(acc.x);
		output[output_tid + 1] = static_cast<unsigned char>(acc.y);
		output[output_tid + 2] = static_cast<unsigned char>(acc.z);

	}

} //end of convGlobal


/*Convolution Wrapper*/

void convolution( const cv::Mat& input, cv::Mat& output ){

	//Calculate the bytes to be transferred
	const int inputBytes = input.step * input.rows;
	const int outputBytes = output.step * output.rows;

	//Instantiate device pointers
	unsigned char *d_input, *d_output;

	//Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input,inputBytes),"CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output,outputBytes),"CUDA Malloc Failed");

	//Calculate required threads and gridSize size to cover the whole image

	//Specify a reasonable blockSize size
	const dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE); //16x16 threads = 256 thread per block

	//Calculate gridSize size to cover the whole image
	const dim3 gridSize((input.cols + blockSize.x - 1)/blockSize.x, (input.rows + blockSize.y - 1)/blockSize.y);

	SAFE_CALL(hipMemcpy(d_input,input.ptr(),inputBytes,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");

	//Launch the convolution kernel
	int radius = 2;
	int weight = 256;

	conv_globalMem<<<gridSize,blockSize>>>(d_input,d_output,input.cols,input.rows,input.step,output.step, radius, weight);

	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");

	//Copy back data from destination  device memory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(),d_output,outputBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

	//Free the device memory
	SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");


}
